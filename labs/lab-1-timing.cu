#include <getopt.h>
#include <sys/time.h>
#include <unistd.h>
#include "utils/tensor.cuh"
#include "ops/op_mm.cuh"
#include "ops/op_elemwise.cuh"

unsigned long long randgen_seed = 0;
void test_matmul(int m, int n, int k, bool on_gpu) {
    Tensor<float> A{m, k, on_gpu};
    op_uniform_init(A);
    Tensor<float> B{k, n, on_gpu};
    op_uniform_init(B);
    Tensor<float> C{m, n, on_gpu};
    op_mm(A, B, C);
    Tensor<float> C2{n, m, on_gpu};
    op_mm(B.transpose(), A.transpose(), C2);
    assert(op_allclose(C2.transpose(), C)); // test transpose
}

int main(int argc, char *argv[]) {
    bool test_gpu = true;
    int test_m = 335, test_n = 587, test_k= 699;
    for (;;) {
        switch (getopt(argc, argv, "s:cm:n:k:")) {
        case 's':
            randgen_seed = atoll(optarg);
            continue;
        case 'c': //cpu testing only
            test_gpu = false;
            continue;
        case 'm':
            test_m = atoi(optarg);
            continue;
        case 'n':
            test_n = atoi(optarg);
            continue;
        case 'k':
            test_k = atoi(optarg);
            continue;
        case -1:
            break;
        }
        break;
    }
    struct timeval start, finish;
    gettimeofday(&start, NULL);
    test_matmul(test_m, test_n, test_k, test_gpu);
    hipDeviceSynchronize();
    gettimeofday(&finish, NULL);
    double t = (finish.tv_sec - start.tv_sec) * 1000000 + (finish.tv_usec - start.tv_usec);
    std::cout << t / 1000 << std::endl;  // ms
    return 0;
}
